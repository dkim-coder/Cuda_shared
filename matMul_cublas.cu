#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "matMul.cuh"

// column major order
#define IDX2C(i,j,ld) (((j)*(ld))+(i))    // i == row, j == column


int cublasMat(const Matrix A, const Matrix B, Matrix C) {
    hipError_t cudaStat;   // cuda_runtime library status
    hipblasStatus_t stat;    // cudlas_v2 library status
    hipblasHandle_t handle;  

    Matrix d_A, d_B, d_C;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size_t size_a = d_A.height * d_A.width * sizeof(double);
    size_t size_b = d_B.height * d_B.width * sizeof(double);
    size_t size_c = d_C.height * d_C.width * sizeof(double);

    int m, n, k;
    m = d_A.height;
    n = d_B.width;
    k = d_A.width;
    
    // memory allocation
    cudaStat = hipMalloc(&d_A.elements, size_a);
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc(&d_B.elements, size_b);
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc(&d_C.elements, size_c);
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed");
        return EXIT_FAILURE;
    }


    // Create handle
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);

        return EXIT_FAILURE;
    }

 
    // set Matrix d_A, d_B
    stat = hipblasSetMatrix(A.height, A.width, sizeof(double), A.elements, A.height, d_A.elements, d_A.height);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download from host to device failed");
        hipblasDestroy(handle);
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
        
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix(B.height, B.width, sizeof(double), B.elements, B.height, d_B.elements, d_B.height);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download from host to device failed");
        hipblasDestroy(handle);
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
        
        return EXIT_FAILURE;
    }
    
    // d_C = d_A * d_B
    double const alpha(1.0);
    double const beta(0.0);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // ???
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B.elements, n, d_A.elements, k, &beta, d_C.elements, n);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("mutiply matrix in device failed");
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
        hipblasDestroy(handle);
        
        return EXIT_FAILURE;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cublas --->>> GPU���� ��İ� ����ð� : % .8f second\n", milliseconds / 1000.);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    
   // Get matrix
    stat = hipblasGetMatrix(d_C.height, d_C.width, sizeof(double), d_C.elements, d_C.height, C.elements, C.height);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download from device to host failed");
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
        hipblasDestroy(handle);

        return EXIT_FAILURE;
    }


    // Free memory on GPU side
    hipblasDestroy(handle);
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    return EXIT_SUCCESS;
}
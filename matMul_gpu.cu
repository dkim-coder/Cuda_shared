#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "matMul.cuh"
#include <stdio.h>
#include <stdlib.h>


__device__ double GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}


__device__ void SetElement(Matrix A, int row, int col, double value)
{
    A.elements[row * A.stride + col] = value;
}


__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    
    return Asub;
}


void matGPU(const Matrix A, const Matrix B, Matrix C)
{
    Matrix d_A, d_B, d_C;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;

    size_t size_a = d_A.height * d_A.width * sizeof(double);
    size_t size_b = d_B.height * d_B.width * sizeof(double);
    size_t size_c = d_C.height * d_C.width * sizeof(double);
    
    // memory allocation
    hipMalloc(&d_A.elements, size_a);
    hipMemcpy(d_A.elements, A.elements, size_a, hipMemcpyHostToDevice);
    hipMalloc(&d_B.elements, size_b);
    hipMemcpy(d_B.elements, B.elements, size_b, hipMemcpyHostToDevice);
    hipMalloc(&d_C.elements, size_c);
    hipMemset(d_C.elements, 0, size_c);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x, (A.height + dimBlock.y - 1) / dimBlock.y);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    MatMulKernel <<<dimGrid, dimBlock >>> (d_A, d_B, d_C);    

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU���� ��İ� ����ð� : % .8f second\n", milliseconds/1000.);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size_c, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int row = threadIdx.y;
    int col = threadIdx.x;
    int x = BLOCK_SIZE * blockCol + col;
    int y = BLOCK_SIZE * blockRow + row;

    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

    double Cvalue = 0.0;

    for (int k = 0; k < (A.width + BLOCK_SIZE - 1) / BLOCK_SIZE; k++) {
        Matrix Asub = GetSubMatrix(A, blockRow, k);
        Matrix Bsub = GetSubMatrix(B, k, blockCol);

        As[row][col] = 0.0;
        Bs[row][col] = 0.0;

        if (k == ((A.width + BLOCK_SIZE - 1) / BLOCK_SIZE - 1)) {
            if (y < A.height && (col < A.width - k * BLOCK_SIZE)) As[row][col] = GetElement(Asub, row, col);
            if (x < B.width && (row < A.width - k * BLOCK_SIZE)) Bs[row][col] = GetElement(Bsub, row, col);
        }
        else {
            if (y < A.height) As[row][col] = GetElement(Asub, row, col);
            if (x < B.width) Bs[row][col] = GetElement(Bsub, row, col);
        }
        __syncthreads();
        
        for (int e = 0; e < BLOCK_SIZE; e++) {
            Cvalue += As[row][e] * Bs[e][col];
        }

        __syncthreads();
    }
    
    if (x < B.width && y < A.height) {
        SetElement(Csub, row, col, Cvalue);
    }
}
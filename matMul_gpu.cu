#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "matMul.cuh"
#include <stdio.h>
#include <stdlib.h>


#define BLOCK_SIZE 3


__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}


__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.elements[row * A.stride + col] = value;
}


__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return Asub;
}


void matGPU(const Matrix A, const Matrix B, Matrix C)
{
    Matrix d_A, d_B, d_C;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;

    size_t size_a = d_A.height * d_A.width * sizeof(float);
    size_t size_b = d_B.height * d_B.width * sizeof(float);
    size_t size_c = d_C.height * d_C.width * sizeof(float);
    
    // memory allocation
    hipMalloc(&d_A.elements, size_a);
    hipMemcpy(d_A.elements, A.elements, size_a, hipMemcpyHostToDevice);
    hipMalloc(&d_B.elements, size_b);
    hipMemcpy(d_B.elements, B.elements, size_b, hipMemcpyHostToDevice);
    hipMalloc(&d_C.elements, size_c);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x, (A.height + dimBlock.y - 1) / dimBlock.y);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU���� ��İ� ����ð� : % .8f second\n", milliseconds/1000.);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size_c, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int row = threadIdx.y;
    int col = threadIdx.x;
    int x = BLOCK_SIZE * blockCol + col;
    int y = BLOCK_SIZE * blockRow + row;

    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    float Cvalue = 0;


    for (int k = 0; k < (A.width + BLOCK_SIZE - 1) / BLOCK_SIZE; ++k) {
        Matrix Asub = GetSubMatrix(A, blockRow, k);
        Matrix Bsub = GetSubMatrix(B, k, blockCol);

        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        __syncthreads();

        if (k == ((A.width + BLOCK_SIZE - 1) / BLOCK_SIZE) - 1) {
            for (int e = 0; e < (A.width - k * BLOCK_SIZE); e++) {
                Cvalue += As[row][e] * Bs[e][col];
            }
        }
        else {
            for (int e = 0; e < BLOCK_SIZE; e++) {
                Cvalue += As[row][e] * Bs[e][col];
            }
        }
        __syncthreads();
    }

    if(x < B.width && y < A.height) SetElement(Csub, row, col, Cvalue);
}